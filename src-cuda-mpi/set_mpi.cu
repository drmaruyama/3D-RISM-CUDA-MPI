#include "hip/hip_runtime.h"
#include <iostream>
#include "rism3d.h"

void RISM3D :: set_mpi () {
  void factor(int, int *);
  MPI_Comm_size(MPI_COMM_WORLD, &procs);
  MPI_Comm_rank(MPI_COMM_WORLD, &myrank);

  int * ip = new int[3];
  factor(procs, ip);
  zprocs = pow(2, (ip[0]  + 1) / 2) * pow(3, (ip[1]  + 1) / 2) 
    * pow(5, (ip[2] + 1) / 2);
  yprocs = procs / zprocs;

  int ycolor = myrank / yprocs;
  int zcolor = myrank % yprocs;
  MPI_Comm_split(MPI_COMM_WORLD, ycolor, 0, &ICOMMY);
  MPI_Comm_split(MPI_COMM_WORLD, zcolor, 0, &ICOMMZ);

  MPI_Comm_rank(ICOMMY, &yrank);
  MPI_Comm_rank(ICOMMZ, &zrank);

  MPI_Bcast(&adswitch, 1, MPI_INT, 0, MPI_COMM_WORLD);
  MPI_Bcast(&lambda, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
}
