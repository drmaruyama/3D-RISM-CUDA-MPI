#include "hip/hip_runtime.h"
#include <iostream>
#include "fft3d.h"

void FFT3D :: initialize (Cell * ce, int procs, int yprocs, int zprocs, 
                          int myrank, MPI_Comm ICY, MPI_Comm ICZ, int devid) {
  __global__ void setkf(double2 * dkf, int nx, int ny, int nz, int, int);
  __global__ void setir(int *, int, int);
  __global__ void setik(int *, int, int);

//  hipSetDevice(devid);

  ngrid = ce -> mgrid;
  volf = ce -> dv;
  volb = 1.0 / ce -> volume;

  nx = ce -> grid[0];
  ny = ce -> grid[1];
  nz = ce -> grid[2];
  nnxy = ce -> xksize;
  nnyy = ce -> yrsize;
  nnyz = ce -> yksize;
  nnzz = ce -> zrsize;
  npuy = yprocs;
  npuz = zprocs;
  ICOMMY = ICY;
  ICOMMZ = ICZ;

  br.x = ce -> grid[0];
  gr.x = ce -> yrsize;
  gr.y = ce -> zrsize;
  bk.x = ce -> xksize;
  gk.x = ce -> yksize;
  gk.y = ce -> grid[2];

  brx.x = ce -> grid[0];
  grx.x = nnyy;
  grx.y = nnzz;
  bry.x = ce -> grid[1];
  gry.x = nnzz;
  gry.y = nnxy;
  brz.x = ce -> grid[2];
  grz.x = nnxy;
  grz.y = nnyz;

  bkx.x = nnyy;
  gkx.x = nnzz;
  gkx.y = ce -> grid[0];
  bky.x = nnzz;
  gky.x = nnxy;
  gky.y = ce -> grid[1];
  bkz.x = nnxy;
  gkz.x = nnyz;
  gkz.y = ce -> grid[2];

  hipHostMalloc(&tmps, ngrid * sizeof(double2));
  hipHostMalloc(&tmpr, ngrid * sizeof(double2));
  hipMalloc(&work, ngrid * sizeof(double2));
  hipMalloc(&dkf, ngrid * sizeof(double2));
  hipMalloc(&dir, ngrid * sizeof(int));
  hipMalloc(&dik, ngrid * sizeof(int));
  hipfftPlan1d(&planx, nx, HIPFFT_Z2Z, nnyy * nnzz);
  hipfftPlan1d(&plany, ny, HIPFFT_Z2Z, nnzz * nnxy);
  hipfftPlan1d(&planz, nz, HIPFFT_Z2Z, nnxy * nnyz);

  setkf <<< gr, br >>> (dkf, ce -> grid[0], ce -> grid[1], ce -> grid[2], 
		        ce -> ystart, ce -> zstart);
  setir <<< gr, br >>> (dir, ce -> ystart, ce -> zstart);
  setik <<< gk, bk >>> (dik, ce -> xks, ce -> yks);
}


void FFT3D :: execute (double2 * da, int key) {
  if (key == - 1) {
    forward(da);
  } else {
    backward(da);
  }
}


void FFT3D :: forward (double2 * da) {
  __global__ void timeirvolf(double2 *, double2 *, int *, double);
  __global__ void timekf(double2 *, double2 *, int *);
  __global__ void ztransf(double2 *, const double2 * __restrict__);
  __global__ void mztransf(double2 *, double2 *, int);

  timekf <<< gr, br >>> (da, dkf, dir);
  hipfftExecZ2Z(planx, da, da, HIPFFT_FORWARD);
  ztransf <<< grx, brx, brx.x * sizeof(double2) >>> (work, da);    

#ifdef GDR
  hipDeviceSynchronize();
  MPI_Alltoall(work, ngrid / npuy, MPI_DOUBLE_COMPLEX, da, ngrid / npuy,
	       MPI_DOUBLE_COMPLEX, ICOMMY);
#else
  hipMemcpy(tmpr, work, ngrid * sizeof(double2), hipMemcpyDefault);
  MPI_Alltoall(tmpr, ngrid / npuy, MPI_DOUBLE_COMPLEX, tmps, ngrid / npuy,
               MPI_DOUBLE_COMPLEX, ICOMMY);
  hipMemcpyAsync(da, tmps, ngrid * sizeof(double2), hipMemcpyDefault);
#endif

  mztransf <<< gkx, bkx >>> (work, da, npuy);
  hipfftExecZ2Z(plany, work, work, HIPFFT_FORWARD);
  ztransf <<< gry, bry, bry.x * sizeof(double2) >>> (da, work);    

#ifdef GDR
  hipDeviceSynchronize();
  MPI_Alltoall(da, ngrid / npuz, MPI_DOUBLE_COMPLEX, work, ngrid / npuz,
	       MPI_DOUBLE_COMPLEX, ICOMMZ);
#else
  hipMemcpy(tmpr, da, ngrid * sizeof(double2), hipMemcpyDefault);
  MPI_Alltoall(tmpr, ngrid / npuz, MPI_DOUBLE_COMPLEX, tmps, ngrid / npuz,
	       MPI_DOUBLE_COMPLEX, ICOMMZ);
  hipMemcpyAsync(work, tmps, ngrid * sizeof(double2), hipMemcpyDefault);
#endif
  mztransf <<< gky, bky >>> (da, work, npuz);
  hipfftExecZ2Z(planz, da, da, HIPFFT_FORWARD);
  ztransf <<< grz, brz, brz.x * sizeof(double2) >>> (work, da);    
  timeirvolf <<< gk, bk >>> (da, work, dik, volf);
}


void FFT3D :: backward (double2 * da) {
  __global__ void timeirvolb(double2 *, int *);
  __global__ void timekb(double2 *, double2 *, double2 *, int *, double);
  __global__ void ztransb(double2 *, const double2 * __restrict__);
  __global__ void mztransb(double2 *, double2 *, int);

  timeirvolb <<< gk, bk >>> (da, dik);
  ztransb <<< grz, brz, brz.x * sizeof(double2) >>> (work, da);    
  hipfftExecZ2Z(planz, work, work, HIPFFT_BACKWARD);
  mztransb <<< grz, brz >>> (da, work, npuz);

#ifdef GDR
  hipDeviceSynchronize();
  MPI_Alltoall(da, ngrid / npuz, MPI_DOUBLE_COMPLEX, work, ngrid / npuz,
	       MPI_DOUBLE_COMPLEX, ICOMMZ);
#else
  hipMemcpy(tmpr, da, ngrid * sizeof(double2), hipMemcpyDefault);
  MPI_Alltoall(tmpr, ngrid / npuz, MPI_DOUBLE_COMPLEX, tmps, ngrid / npuz,
	       MPI_DOUBLE_COMPLEX, ICOMMZ);
  hipMemcpyAsync(work, tmps, ngrid * sizeof(double2), hipMemcpyDefault);
#endif

  ztransb <<< gry, bry, bry.x * sizeof(double2) >>> (da, work);    
  hipfftExecZ2Z(plany, da, da, HIPFFT_BACKWARD);
  mztransb <<< gry, bry >>> (work, da, npuy);

#ifdef GDR
  hipDeviceSynchronize();
  MPI_Alltoall(work, ngrid / npuy, MPI_DOUBLE_COMPLEX, da, ngrid / npuy,
	       MPI_DOUBLE_COMPLEX, ICOMMY);
#else
  hipMemcpy(tmpr, work, ngrid * sizeof(double2), hipMemcpyDefault);
  MPI_Alltoall(tmpr, ngrid / npuy, MPI_DOUBLE_COMPLEX, tmps, ngrid / npuy,
	       MPI_DOUBLE_COMPLEX, ICOMMY);
  hipMemcpyAsync(da, tmps, ngrid * sizeof(double2), hipMemcpyDefault);
#endif

  ztransb <<< grx, brx, brx.x * sizeof(double2) >>> (work, da);    
  hipfftExecZ2Z(planx, work, work, HIPFFT_BACKWARD);
  timekb <<< gr, br >>> (da, work, dkf, dir, volb);
}
