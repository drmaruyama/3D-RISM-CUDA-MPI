#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include "rism3d.h"

void RISM3D :: initialize_g() {
  __global__ void setg(double3 * dgv, double * dg2, 
		       double bx, double by, double bz,
		       int nx, int ny, int nz, int xx, int yy);
  void index(double * &, int * &, int);

  indga = new int[ce -> mgrid];
  double * g2 = new double[ce -> mgrid];
  int * indg2 = new int[ce -> mgrid];

  double * dg2;
  hipMalloc(&dgv, ce -> mgrid * sizeof(double3));
  hipMalloc(&dg2, ce -> mgrid * sizeof(double));

  setg <<< gk, bk >>> (dgv, dg2, ce -> box[0], ce -> box[1], ce -> box[2],
		      ce -> grid[0], ce -> grid[1], ce -> grid[2], 
		      ce -> xks, ce -> yks);

  hipMemcpyAsync(g2, dg2, ce -> mgrid * sizeof(double), hipMemcpyDefault);
  thrust::device_vector<int> indg(ce -> mgrid);
  thrust::device_ptr<double> dg2_ptr(dg2);
  thrust::sequence(indg.begin(), indg.end());
  thrust::sort_by_key(dg2_ptr, dg2_ptr + ce -> mgrid, indg.begin());  
  thrust::copy(indg.begin(), indg.end(), indg2);
  //  index(g2, indg2, ce -> mgrid);

  double ga2o = - 1.0;
  nga = 0;

  for (int igk = 0; igk < ce -> mgrid; ++igk) {
    int igs = indg2[igk];
    double ga2 = g2[igs];
    if (ga2 > ga2o) {
      ++nga;
      ga . push_back (sqrt(ga2));
      ga2o = ga2;
    }
    indga[igs] = nga - 1;
  }

  hipFree(dg2);
  delete[] g2;
  delete[] indg2;
}


__global__ void setg(double3 * dgv, double * dg2, 
		     double bx, double by, double bz,
		     int nx, int ny, int nz, int xx, int yy) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  dgv[ip].x = 2.0 * M_PI * (threadIdx.x + xx - nx / 2.0 + 0.5) / bx;
  dgv[ip].y = 2.0 * M_PI * (blockIdx.x + yy - ny / 2.0 + 0.5) / by;
  dgv[ip].z = 2.0 * M_PI * (blockIdx.y - nz / 2.0 + 0.5) / bz;
  dg2[ip] = dgv[ip].x * dgv[ip].x + dgv[ip].y * dgv[ip].y 
    + dgv[ip].z * dgv[ip].z;
}
