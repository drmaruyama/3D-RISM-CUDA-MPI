#include <iostream>
#include <fstream>
#include <unistd.h>
#include "rism3d.h"

int main (int argc, char * argv[]) {
  RISM3D * system;
  int ch, cu, dn;
  string input;
  string structure;

  system = new RISM3D;

  while ((ch = getopt(argc, argv, "c:i:s:l:e:")) != -1) {
    switch (ch){
    case 'c':
      cu = atoi(optarg);
      break;
    case 'i':
      input = optarg;
      break;
    case 's':
      structure = optarg;
      break;
    case 'l':
      system -> set_ad (atof(optarg), 1);
      break;
    case 'e':
      system -> set_ad (atof(optarg), 2);
      break;
    }
  }

  if (input.empty() || structure.empty()) {
    if (argv[optind] == NULL) {
      cout << "No input file!" << endl;
      return (1);
    }
    input = argv[optind];
  }

#ifdef OPENMPI
  dn = atoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK"));
#endif
#ifdef MVAPICH
  dn = atoi(getenv("MV2_COMM_WORLD_LOCAL_RANK"));
#endif

  hipSetDevice(dn);

  MPI_Init(&argc, &argv);
  //  int provided;
  //  MPI_Init_thread(&argc, &argv, MPI_THREAD_FUNNELED, &provided);

  system -> initialize(input, structure, dn);
  system -> iterate();
  system -> output();    

  MPI_Finalize();
  return(0);
}
