#include <iostream>
#include <fstream>
#include <unistd.h>
#include "rism3d.h"

int main (int argc, char * argv[]) {
  RISM3D * system;
  int ch, cu, dn;

  system = new RISM3D;

  while ((ch = getopt(argc, argv, "c:l:e:")) != -1) {
    switch (ch){
    case 'c':
      cu = atoi(optarg);
      break;
    case 'l':
      system -> set_ad (atof(optarg), 1);
      break;
    case 'e':
      system -> set_ad (atof(optarg), 2);
      break;
    }
  }
  if (argc == 1) {
    cout << "No parameter file!" << endl ;
    return (1) ;
  }

#ifdef OPENMPI
  dn = atoi(getenv("OMPI_COMM_WORLD_LOCAL_RANK"));
#endif
#ifdef MVAPICH
  dn = atoi(getenv("MV2_COMM_WORLD_LOCAL_RANK"));
#endif

  hipSetDevice(dn);

  MPI_Init(&argc, &argv);
  //  int provided;
  //  MPI_Init_thread(&argc, &argv, MPI_THREAD_FUNNELED, &provided);

  system -> initialize(argv[optind], dn);
  system -> iterate();
  system -> output();    

  MPI_Finalize();
  return(0);
}
