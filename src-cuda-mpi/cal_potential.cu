#include "hip/hip_runtime.h"
#include "rism3d.h"

void RISM3D :: cal_potential() {
  __global__ void set_du(double * du, double * de, double q);
#ifdef TEST
  double t[5];
  t[0] = MPI_Wtime();
#endif

  cal_LJ();
#ifdef TEST
  t[1] = MPI_Wtime();
#endif
  cal_Coulomb();
#ifdef TEST
  t[2] = MPI_Wtime();
#endif

  for (int iv = 0; iv < sv -> natv; ++iv) {
    set_du <<< gr, br >>> (du + (iv * ce -> mgrid), de, sv -> qv[iv]);
  }
#ifdef TEST
  t[3] = MPI_Wtime();
#endif
//  hipFree(dgv);
//  hipFree(de);
#ifdef TEST
  t[4] = MPI_Wtime();
  if (myrank == 0) {
    printf("LJ          :\t%lf sec.\n", t[1] - t[0]);
    printf("Coulomb     :\t%lf sec.\n", t[2] - t[1]);
    printf("Summation   :\t%lf sec.\n", t[3] - t[2]);
//    printf("hipFree    :\t%lf sec.\n", t[4] - t[3]);
  }
#endif
}


__global__ void set_du(double * du, double * de, double q) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x 
    + blockIdx.y * blockDim.x * gridDim.x;
  du[ip] += q * de[ip];
}
