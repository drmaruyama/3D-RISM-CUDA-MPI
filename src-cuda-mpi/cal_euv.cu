#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include "rism3d.h"

__constant__ double3 dv;
__constant__ int3 grid;

void RISM3D :: cal_euv (double * & e) {
  __global__ void euv(double * ds, double2 * dguv, double * dsig,
                      double * deps,  double3 * dr, double * qu, double gv,
                      int natu, int iv, int iu, int yy, int zz);

  int ng = ce -> mgrid;		      

  hipMemcpyToSymbol(HIP_SYMBOL(dv), ce -> dr, sizeof(double3));
  hipMemcpyToSymbol(HIP_SYMBOL(grid), ce -> grid, sizeof(int3));

  for (size_t iv = 0; iv < sv -> natv; ++iv) {
    for (size_t iu = 0; iu < su -> num; ++iu) {
      euv <<< gr, br, br.x * sizeof(double) >>>
        (ds, dguv + iv * ng, dsig, deps, su -> dr, su -> dq, sv -> qv[iv], 
         su -> num, iv, iu, ce -> ystart, ce -> zstart);
	thrust::device_ptr<double> ds_ptr(ds);
        double s = thrust::reduce(ds_ptr, ds_ptr + (gr.x * gr.y));
        e[iu * sv -> natv + iv] = s * sv -> rhov[iv];
    }
  }
}

__global__ void euv(double * ds, double2 * dguv, double * dsig,
                    double * deps,  double3 * dr, double * qu,
                    double qv, int natu, int iv, int iu, int yy, int zz) {
  extern __shared__ double sdata[];
  const double cc = hartree * bohr * avogadoro;

  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  int iuv = iu + iv * natu;

  double dx = ((int)threadIdx.x - grid.x / 2) * dv.x - dr[iu].x;
  double dy = ((int)blockIdx.x + yy - grid.y / 2) * dv.y - dr[iu].y;
  double dz = ((int)blockIdx.y + zz - grid.z / 2) * dv.z - dr[iu].z;
  double r2 = dx * dx + dy * dy + dz * dz;
  double r1 = sqrt(r2);

  if (r1 < dsig[iuv] * 0.5) {
    sdata[threadIdx.x] = 0.0;
  } else {
    double rs2i = dsig[iuv] * dsig[iuv] / r2;
    double rs6i = rs2i * rs2i * rs2i;
    double ulj = deps[iuv] * 4.0 * rs6i * ( rs6i - 1.0) * dguv[ip].x;
    double uco = qu[iu] * qv / r1 * cc * dguv[ip].x;
    sdata[threadIdx.x] = ulj + uco;
  }
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      sdata[threadIdx.x] += sdata[threadIdx.x + s];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    ds[blockIdx.x + blockIdx.y * gridDim.x] = sdata[0];
  }
}



/*
__global__ void ehnc(double * ds, double2 * dhuv, double * dt) {
  extern __shared__ double sdata[];

  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;

  sdata[threadIdx.x] = dhuv[ip].x * 0.5 * dt[ip] - (dhuv[ip]. x - dt[ip]);
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (threadIdx.x < s) {
      sdata[threadIdx.x] += sdata[threadIdx.x + s];
    }
    __syncthreads();
  }
  if (threadIdx.x < 32) {
    volatile double *smem = sdata;
    smem[threadIdx.x] += smem[threadIdx.x + 32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 2];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 1];
  }
  if (threadIdx.x == 0) ds[blockIdx.x + blockIdx.y * gridDim.x] = sdata[0];
}
*/