#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include "rism3d.h"

template <typename T>
struct square {
  __host__ __device__ T operator()(const T &x) const { 
    return x * x;
  }
};


double RISM3D :: cal_rms () {
  __global__ void reduce0(double * ds, double * dtr);

  square<double> uop;
  thrust::plus<double> bop;
  thrust::device_ptr<double> dtr_ptr(dtr);
  double rms = thrust::transform_reduce(dtr_ptr, dtr_ptr 
				+ sv -> natv * ce -> mgrid, uop, 0.0, bop);
  double rms00;
  MPI_Allreduce(&rms, &rms00, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
  rms = sqrt (rms00 / (ce -> ngrid * sv -> natv));
  return rms;
}

__global__ void reduce0(double * ds, double * dtr) {
  extern __shared__ double sdata[];

  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;

  sdata[threadIdx.x] = dtr[ip] * dtr[ip];
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (threadIdx.x < s) {
      sdata[threadIdx.x] += sdata[threadIdx.x + s];
    }
    __syncthreads();
  }
  if (threadIdx.x < 32) {
    volatile double *smem = sdata;
    smem[threadIdx.x] += smem[threadIdx.x + 32];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 16];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 8];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 4];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 2];
    __syncwarp();
    smem[threadIdx.x] += smem[threadIdx.x + 1];
  }
  if (threadIdx.x == 0) ds[blockIdx.x + blockIdx.y * gridDim.x] = sdata[0];
}

