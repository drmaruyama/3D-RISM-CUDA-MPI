#include "hip/hip_runtime.h"
#include <iostream>
#include "rism3d.h"

void RISM3D :: cal_Coulomb () {
  __global__ void coulomb(double * de, double * dfr,
			  double3 * dru, double * dqu,
			  double dx, double dy, double dz,
			  int nx, int ny, int nz, int natu, int ystart, int zstart);
  __global__ void fk(double2 *, double3 *, double3 *, double *, int);
  __global__ void betar(double * de, double * dfr, double ubeta);
  __global__ void betak(double2 * dfk, double ubeta);

  if (myrank == 0) {
    cout << "synthesizing solute Coulomb potential ..." << endl;
  }
  hipMalloc(&de, ce -> mgrid * sizeof(double));
  hipMalloc(&dfr, ce -> mgrid * sizeof(double));
  hipMalloc(&dfk, ce -> mgrid * sizeof(double2));
  hipMemsetAsync(de, 0.0, ce -> mgrid * sizeof(double));
  hipMemsetAsync(dfr, 0.0, ce -> mgrid * sizeof(double));
  hipMemsetAsync(dfk, 0.0, ce -> mgrid * sizeof(double2));

  if (adswitch != 1) {
    coulomb <<< gr, br >>> (de, dfr, su -> dr, su -> dq,
      			    ce -> dr[0], ce -> dr[1], ce -> dr[2], 
      			    ce -> grid[0], ce -> grid[1], ce -> grid[2],
      			    su -> num, ce -> ystart, ce -> zstart);

    fk <<< gk, bk >>> (dfk, dgv, su -> dr, su -> dq, su -> num);

    double lambda2 = 1.0;
    if (adswitch == 2) lambda2 = lambda;
    double ubeta = hartree * bohr / (boltzmann * sv -> temper) * lambda2;
    betar <<< gr, br >>> (de, dfr, ubeta);
    betak <<< gk, bk >>> (dfk, ubeta);
  }
} 


__global__ void coulomb(double * de, double * dfr,
                        double3 * dru, double * dqu,
                        double bx, double by, double bz,
                        int nx, int ny, int nz, int natu, int yy, int zz) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  double rx = ((int)threadIdx.x - nx / 2) * bx;
  double ry = ((int)blockIdx.x + yy - ny / 2) * by;
  double rz = ((int)blockIdx.y + zz - nz / 2) * bz;
  for (int iu = 0; iu < natu; ++iu) {
    double delx = rx - dru[iu].x;
    double dely = ry - dru[iu].y;
    double delz = rz - dru[iu].z;
    double ra = sqrt(delx * delx + dely * dely + delz * delz) ;
    if (ra >= 1.0e-5) {
      double qr = dqu[iu] / ra ;
      de[ip] += qr ;
      dfr[ip] += qr * (1 - exp(- ra)) ;
    } else {
      dfr[ip] += dqu[iu] ;
    }
  }
}


__global__ void fk(double2 * dfk, double3 * dgv, double3 * dru, double * dqu,
		   int natu) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  double rk2 = dgv[ip].x * dgv[ip].x
    + dgv[ip].y * dgv[ip].y + dgv[ip].z * dgv[ip].z;
  double rk4i = 1.0 / (rk2 * (rk2 + 1.0));
  for (int iu = 0; iu < natu; ++iu) {
    double ruk = dgv[ip].x * dru[iu].x 
      + dgv[ip].y * dru[iu].y + dgv[ip].z * dru[iu].z;
    double tmp = 4.0 * M_PI * dqu[iu] * rk4i;
    dfk[ip].x += tmp * cos(ruk);
    dfk[ip].y -= tmp * sin(ruk);
  }
}


__global__ void betar(double * de, double * dfr, double ubeta) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  de[ip] *= ubeta;
  dfr[ip] *= ubeta;
}


__global__ void betak(double2 * dfk, double ubeta) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  dfk[ip].x *= ubeta;
  dfk[ip].y *= ubeta;
}
