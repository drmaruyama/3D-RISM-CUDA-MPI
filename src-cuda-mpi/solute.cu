#include "hip/hip_runtime.h"
#include "solute.h"

void Solute :: init(int n) {
  num = n;
  hipHostMalloc(&q, num * sizeof(double));
  hipHostMalloc(&r, num * 3 * sizeof(double));
//  hipHostMalloc(&sig, num * sizeof(double));
//  hipHostMalloc(&eps, num * sizeof(double));
//  q = new double[num];
  sig = new double[num];
  eps = new double[num];
//  r = new double[num * 3];
}

void Solute :: setup_mpi() {
  MPI_Bcast(q, num, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(sig, num, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(eps, num, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(r, num * 3, MPI_DOUBLE, 0, MPI_COMM_WORLD);
}

void Solute :: setup_cuda() {
  hipMalloc(&dq, num * sizeof(double));
  hipMalloc(&dr, num * sizeof(double3));
  hipMemcpy(dq, q, num * sizeof(double), hipMemcpyDefault);
  hipMemcpy(dr, r, num * sizeof(double3), hipMemcpyDefault);
}
