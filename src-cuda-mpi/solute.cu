#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <limits>
#include "solute.h"

void Solute :: init(int n) {
  num = n;
  hipHostMalloc(&q, num * sizeof(double));
  hipHostMalloc(&r, num * 3 * sizeof(double));
//  hipHostMalloc(&sig, num * sizeof(double));
//  hipHostMalloc(&eps, num * sizeof(double));
//  q = new double[num];
  sig = new double[num];
  eps = new double[num];
//  r = new double[num * 3];
}

double * Solute :: centering() {
  double xmin, ymin, zmin, xmax, ymax, zmax;
  double * shift;
  shift = new double[3];

  xmin = ymin = zmin = std::numeric_limits<double>::max();
  xmax = ymax = zmax = std::numeric_limits<double>::lowest();

  for (int n = 0; n < num; ++n) {
    int i = n * 3;
    if (xmin > r[i]) xmin = r[i];
    if (ymin > r[i + 1]) ymin = r[i + 1];
    if (zmin > r[i + 2]) zmin = r[i + 2];
    if (xmax < r[i]) xmax = r[i];
    if (ymax < r[i + 1]) ymax = r[i + 1];
    if (zmax < r[i + 2]) zmax = r[i + 2];
  }

  shift[0] = round(- (xmax - xmin) / 2 - xmin);
  shift[1] = round(- (ymax - ymin) / 2 - ymin);
  shift[2] = round(- (zmax - zmin) / 2 - zmin);

  for (int n = 0; n < num; ++n) {
    int i = n * 3;
    r[i] += shift[0];
    r[i + 1] += shift[1];
    r[i + 2] += shift[2];
  }

 return shift;
}

void Solute :: setup_mpi() {
  MPI_Bcast(q, num, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(sig, num, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(eps, num, MPI_DOUBLE, 0, MPI_COMM_WORLD);
  MPI_Bcast(r, num * 3, MPI_DOUBLE, 0, MPI_COMM_WORLD);
}

void Solute :: setup_cuda() {
  hipMalloc(&dq, num * sizeof(double));
  hipMalloc(&dr, num * sizeof(double3));
  hipMemcpy(dq, q, num * sizeof(double), hipMemcpyDefault);
  hipMemcpy(dr, r, num * sizeof(double3), hipMemcpyDefault);
}
