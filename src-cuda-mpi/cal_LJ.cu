#include "hip/hip_runtime.h"
#include <iostream>
#include "rism3d.h"

void RISM3D :: cal_LJ() {
  __global__ void LJ(double * du, double * dsig, double * deps, double3 * dru,
		     double cut2, double ikbt, double bx, double by, double bz,
		     int nx, int ny, int nz, int natu, int iv, int ystart, int zstart);
  __global__ void time_iKbT(double * du, double ikbt);

//  const double cut = 1.0e-2;
  const double cut = 2.0e-3;
  const double cut2 = cut * cut;

  if (myrank == 0) {
    cout << "tabulating solute Lennard-Jones potential ..." << endl;
  }

  hipMalloc(&du, ce -> mgrid * sv -> natv * sizeof(double));
  hipMalloc(&dsig, su -> num * sv -> natv * sizeof(double));
  hipMalloc(&deps, su -> num * sv -> natv * sizeof(double));
  hipMemset(du, 0.0, ce -> mgrid * sv -> natv * sizeof(double));

  hipHostMalloc(&siguv, su -> num * sv -> natv * sizeof(double));
  hipHostMalloc(&epsuv, su -> num * sv -> natv * sizeof(double));
//  siguv = new double[su -> num * sv -> natv];
//  epsuv = new double[su -> num * sv -> natv];

  double lambda1;
  if (adswitch == 1) {
    lambda1 = lambda;
  } else {
    lambda1 = 1.0;
  }

  for (int iv = 0; iv < sv -> natv; ++iv) {
#pragma omp parallel for
    for (int iu = 0; iu < su -> num; ++iu) {
      int ip = iu + su -> num * iv;
      siguv[ip] = (su -> sig[iu] + sv -> sigv[iv]) * 0.5 * lambda1;
      epsuv[ip] = sqrt (su -> eps[iu] * sv -> epsv[iv] * kcal2J);
    }
  }

  hipMemcpy(dsig, siguv, su -> num * sv -> natv * sizeof(double),
	     hipMemcpyDefault);
  hipMemcpy(deps, epsuv, su -> num * sv -> natv * sizeof(double),
	     hipMemcpyDefault);

  double iKbT = 1.0 / (avogadoro * boltzmann * sv -> temper);
  for (int iv = 0; iv < sv -> natv; ++iv) {
    LJ <<< gr, br >>> (du + (iv * ce -> mgrid), dsig, deps, su -> dr, 
		      cut2, iKbT, ce -> dr[0], ce -> dr[1], ce -> dr[2], 
		      ce -> grid[0], ce -> grid[1], ce -> grid[2], 
		      su -> num, iv, ce -> ystart, ce -> zstart);
  }
}

__global__ void LJ(double * du, double * dsig, double * deps, double3 * dru,
                   double cut2, double ikbt, double bx, double by, double bz,
                   int nx, int ny, int nz, int natu, int iv, int yy, int zz) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  double rx = ((int)threadIdx.x - nx / 2) * bx;
  double ry = ((int)blockIdx.x + yy - ny / 2) * by;
  double rz = ((int)blockIdx.y + zz - nz / 2) * bz;
  for (int iu = 0; iu < natu; ++iu) {
    int iuv = iu + natu * iv;
    double dx = rx - dru[iu].x;
    double dy = ry - dru[iu].y;
    double dz = rz - dru[iu].z;
    double r2 = dx * dx + dy * dy + dz * dz ;

//    if (r2 < cut2) r2 = cut2;
//    double irs2 = dsig[iuv] * dsig[iuv] / r2;
//    double irs6 = irs2 * irs2 * irs2;

    double rs2 = r2 / (dsig[iuv] * dsig[iuv]);
    if (rs2 < cut2) rs2 = cut2;
    double irs6 = 1.0 / (rs2 * rs2 * rs2);
    du[ip] += deps[iuv] * 4.0 * irs6 * (irs6 - 1.0);
  }
  du[ip] *= ikbt;
}
