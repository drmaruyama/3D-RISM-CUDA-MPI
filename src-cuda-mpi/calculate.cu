#include "hip/hip_runtime.h"
#include <iostream>
#include "rism3d.h"

void RISM3D :: calculate (double & ft) {
  __global__ void kh(double * dtr, double * dt, double * du);
  __global__ void hnc(double * dtr, double * dt, double * du);
  __global__ void trm1mt(double2 * dguv, double * dtr, double * dt,
                         double * dfr, double qv);
  __global__ void mqvfk(double2 * dguv, double2 * dfk, double qv);
  __global__ void oz(double2 * dhuv, double2 * dguv, double * dx, int natv);
  __global__ void tr(double2 * dguv, double * dtr, double2 * dhuv);

  int ng = ce -> mgrid;

  if (clos == 0) {
    for (int iv = 0; iv < sv -> natv; ++iv) {
      kh <<< gr, br >>> (dtr + (iv * ng), dt + (iv * ng), du + (iv * ng));
    }
  } else if (clos == 1) {
    for (int iv = 0; iv < sv -> natv; ++iv) {
      hnc <<< gr, br >>> (dtr + (iv * ng), dt + (iv * ng), du + (iv * ng));
    }
  } 

  for (int iv = 0; iv < sv -> natv; ++iv) {
    trm1mt <<< gr, br >>> (dguv + (iv * ng), dtr + (iv * ng),
		 	  dt + (iv * ng), dfr, sv -> qv[iv]);
  }

#ifdef TEST
  double fs, fe;
  fs = MPI_Wtime();
#endif
  for (int iv = 0; iv < sv -> natv; ++iv) {
    fft -> execute(dguv + (iv * ng), - 1);
  }
#ifdef TEST
  fe = MPI_Wtime();
  ft += fe - fs;
#endif

  for (int iv = 0; iv < sv -> natv; ++iv) {
    mqvfk <<< gk, bk >>> (dguv + (iv * ng), dfk, sv -> qv[iv]);
  }

  for (int iv = 0; iv < sv -> natv; ++iv) {
    oz <<< gk, bk >>> (dhuv + (iv * ng), dguv,
		      sv -> dx + (iv * sv -> natv * ng), sv -> natv);
  }

#ifdef TEST
  fs = MPI_Wtime();
#endif
  for (int iv = 0; iv < sv -> natv; ++iv) {
    fft -> execute(dhuv + (iv * ng), 1);
  }
#ifdef TEST
  fe = MPI_Wtime();
  ft += fe - fs;
#endif

  for (int iv = 0; iv < sv -> natv; ++iv) {
    tr <<< gr, br >>> (dguv + (iv * ng), dtr + (iv * ng), dhuv + (iv * ng));
  }
} 
