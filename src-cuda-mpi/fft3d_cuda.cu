
#include <hip/hip_runtime.h>
__global__ void mztransf(double2 * v, double2 * da, int np) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  int nn = gridDim.x * gridDim.y / np;  
  int yz = blockIdx.x + blockIdx.y * gridDim.x;
  int y = yz / nn;
  int z = yz % nn;
  unsigned int jp = threadIdx.x + y * blockDim.x
    + z * blockDim.x * np;
  v[jp].x = da[ip].x;
  v[jp].y = da[ip].y;
}


__global__ void mztransb(double2 * v, double2 * da, int np) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  int nx = blockDim.x / np;
  int x = threadIdx.x % nx;
  int z = threadIdx.x / nx;
  unsigned int jp = x + blockIdx.x * nx + blockIdx.y * nx * gridDim.x
    + z * nx * gridDim.x * gridDim.y;
  v[jp].x = da[ip].x;
  v[jp].y = da[ip].y;
}


__global__ void ztransf(double2 * v, const double2 * __restrict__ da) {
  extern __shared__ double2 sdata[];

  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  unsigned int jp = blockIdx.x + blockIdx.y * gridDim.x
    + threadIdx.x * gridDim.x * gridDim.y;
  sdata[threadIdx.x].x = da[ip].x;
  sdata[threadIdx.x].y = da[ip].y;
  v[jp].x = sdata[threadIdx.x].x; 
  v[jp].y = sdata[threadIdx.x].y;
}


__global__ void ztransb(double2 * v, const double2 * __restrict__ da) {
  extern __shared__ double2 sdata[];

  unsigned int ip = blockIdx.x + blockIdx.y * gridDim.x
    + threadIdx.x * gridDim.x * gridDim.y;
  unsigned int jp = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  sdata[threadIdx.x].x = da[ip].x;
  sdata[threadIdx.x].y = da[ip].y;
  v[jp].x = sdata[threadIdx.x].x; 
  v[jp].y = sdata[threadIdx.x].y;
}


__global__ void timekf(double2 * da, double2 * dkf, int * dir) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  double tmpr = da[ip].x * dir[ip];
  double tmpi = da[ip].y * dir[ip];
  da[ip].x = tmpr * dkf[ip].x - tmpi * dkf[ip].y;
  da[ip].y = tmpi * dkf[ip].x + tmpr * dkf[ip].y;
}


__global__ void timekb(double2 * da, double2 * v, double2 * dkf,
		       int * dir, double vol) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  double tmpr = v[ip].x * dir[ip] * vol;
  double tmpi = v[ip].y * dir[ip] * vol;
  da[ip].x = tmpr * dkf[ip].x + tmpi * dkf[ip].y;
  da[ip].y = tmpi * dkf[ip].x - tmpr * dkf[ip].y;
}


__global__ void timeirvolf(double2 * da, double2 * v,
			   int * dik, double vol) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  da[ip].x = v[ip].x * dik[ip] * vol;
  da[ip].y = v[ip].y * dik[ip] * vol;
}


__global__ void timeirvolb(double2 * da, int * dik) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  da[ip].x = da[ip].x * dik[ip];
  da[ip].y = da[ip].y * dik[ip];
}


__global__ void setkf(double2 * dkf, int nx, int ny, int nz, int yy, int zz) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  int x = threadIdx.x - nx / 2.0;
  int y = blockIdx.x + yy - ny / 2.0;
  int z = blockIdx.y + zz - nz / 2.0;
  double dkx = M_PI / nx;
  double dky = M_PI / ny;
  double dkz = M_PI / nz;
  double dkr = dkx * x + dky * y + dkz * z;
  dkf[ip].x = cos(dkr);
  dkf[ip].y = - sin(dkr);
}


__global__ void setir(int * dir, int yy, int zz) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  if ((threadIdx.x + blockIdx.x + blockIdx.y + yy + zz) % 2 == 0) {
    dir[ip] = 1;
  } else {
    dir[ip] = -1;
  }
}


__global__ void setik(int * dik, int xx, int yy) {
  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  if ((threadIdx.x + blockIdx.x + blockIdx.y + xx + yy) % 2 == 0) {
    dik[ip] = 1;
  } else {
    dik[ip] = -1;
  }
}
