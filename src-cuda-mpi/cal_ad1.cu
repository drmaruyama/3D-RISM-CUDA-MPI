#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include "rism3d.h"

__constant__ double3 dv;
__constant__ int3 grid;

void RISM3D :: cal_ad1(double * & du) {
  __global__ void ad1(double * ds, double2 * dguv, double * dsig, 
		       double * deps,  double3 * dr, double * qu,
		       double qv, int natu, int iv, int iu, int yy, int zz, 
		       double lambda);

  int ng = ce -> mgrid;

  hipMemcpyToSymbol(HIP_SYMBOL(dv), ce -> dr, sizeof(double3));
  hipMemcpyToSymbol(HIP_SYMBOL(grid), ce -> grid, sizeof(int3));

  double * ds;
  hipMalloc(&ds, gr.x * gr.y * sizeof(double));

#pragma omp parallel for
  for (int iu = 0; iu < su -> num; ++iu) {
    du[iu] = 0.0;
  }

  for (int iv = 0; iv < sv -> natv; ++iv) {
    for (int iu = 0; iu < su -> num; ++iu) {
      ad1 <<< gr, br, br.x * sizeof(double) >>> 
	(ds, dguv + (iv * ng), dsig, deps, su -> dr, su -> dq, sv -> qv[iv], 
	 su -> num, iv, iu, ce -> ystart, ce -> zstart, lambda);

      thrust::device_ptr<double> ds_ptr(ds);
      double s = thrust::reduce(ds_ptr, ds_ptr + gr.x * gr.y);
      du[iu] += s * sv -> rhov[iv];
    }
  }
  hipFree(ds);
}

__global__ void ad1(double * ds, double2 * dguv, double * dsig, 
		   double * deps,  double3 * dr, double * qu,
                   double qv, int natu, int iv, int iu, int yy, int zz, 
 		   double lambda) {
  extern __shared__ double sdata[];
  const double cc = hartree * bohr * avogadoro;

  unsigned int ip = threadIdx.x + blockIdx.x * blockDim.x
    + blockIdx.y * blockDim.x * gridDim.x;
  int iuv = iu + iv * natu;

  double dx = ((int)threadIdx.x - grid.x / 2) * dv.x - dr[iu].x;
  double dy = ((int)blockIdx.x + yy - grid.y / 2) * dv.y - dr[iu].y;
  double dz = ((int)blockIdx.y + zz - grid.z / 2) * dv.z - dr[iu].z;
  double r2 = dx * dx + dy * dy + dz * dz;
  double r1 = sqrt(r2);

  if (r1 < dsig[iuv] * lambda * 0.5) {
    sdata[threadIdx.x] = 0.0;
  } else {
    double rs2i = dsig[iuv] * dsig[iuv] / r2;
    double rs6i = rs2i * rs2i * rs2i;
    double ulj = deps[iuv] * 24.0 * rs6i * (2.0 * rs6i - 1.0) / lambda
      * dguv[ip].x;
    sdata[threadIdx.x] = ulj;
  }
  __syncthreads();

  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      sdata[threadIdx.x] += sdata[threadIdx.x + s];
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    ds[blockIdx.x + blockIdx.y * gridDim.x] = sdata[0];
  }
}
