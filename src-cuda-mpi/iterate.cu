#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include "rism3d.h"
#include "extension.h"

void RISM3D :: iterate() {
  void alloc2D (vector <double *> &, int, int);
  void calloc2D (vector <complex <double> *> &, int, int);
#ifdef TEST
  double t[5];
  double ft = 0.0;
  double mt = 0.0;
  t[0] = MPI_Wtime();
#endif

  calloc2D (guv, sv -> natv, ce -> mgrid);
  calloc2D (huv, sv -> natv, ce -> mgrid);
  alloc2D (tuv, sv -> natv, ce -> mgrid);

  hipMalloc(&dguv, ce -> mgrid * sv -> natv * sizeof(double2));
  hipMalloc(&dhuv, ce -> mgrid * sv -> natv * sizeof(double2));
  hipMalloc(&dt, ce -> mgrid * sv -> natv * sizeof(double));
  hipMalloc(&dtr, ce -> mgrid * sv -> natv * sizeof(double));
  hipMalloc(&ds, ce -> grid[1] * ce -> zrsize * sizeof(double));

  ifstream in_file ;
  in_file.open((fname + exttuv).c_str());
  bool saved = in_file.is_open();
  in_file.close();

  if (saved) {
    read_tuv();
  } else {
    initialize_tuv();
  }
#ifdef TEST
  t[1] = MPI_Wtime();
#endif
  ma -> initialize (ce, sv);
  fft -> initialize (ce, procs, yprocs, zprocs, myrank, ICOMMY, ICOMMZ, devid);
#ifdef TEST
  t[2] = MPI_Wtime();
#endif

  if (myrank == 0) cout << "relaxing 3D UV RISM:" << endl;
  bool conver = false;
  bool diverge = false;
  for (int istep = 1; istep <= co -> maxstep; ++istep) {
    calculate(ft);
    double rms = cal_rms ();
    diverge = !isfinite(rms);
    if (diverge) {
      break;
    }
    if (rms <= co -> convergence) {
      conver = true;
    } else {
      ma -> calculate (dt, dtr, mt);
    }
    if (myrank == 0) {
      cout << " Step = " << istep << " Reside = " << rms << endl;
    }
    if (co -> ksave > 0 && istep % co -> ksave == 0) {
      write_tuv();
    }
    if (conver) {
      if (co -> ksave != 0) {
	write_tuv();
      }
      break;
    }
  }
  if (diverge) {
    if (myrank == 0) {
      cout << "Calculation diverged." << endl;
    }
  } else if (!conver) {
    if (myrank == 0) {
      cout << "3D UV RISM: reached limit # of relaxation steps: "
	   << co -> maxstep << endl;
    }
  }
#ifdef TEST
  t[3] = MPI_Wtime();
#endif
  for (int iv = 0; iv < sv -> natv; ++iv) {
    hipMemcpyAsync(huv[iv], dhuv + (iv * ce -> mgrid), 
	       ce -> mgrid * sizeof(double2), hipMemcpyDefault);
    hipMemcpyAsync(guv[iv], dguv + (iv * ce -> mgrid), 
	       ce -> mgrid * sizeof(double2), hipMemcpyDefault);
  }
  delete ma;
  delete fft;
#ifdef TEST
  t[4] = MPI_Wtime();
  if (myrank == 0) {
    printf("Init Tuv    :\t%lf sec.\n", t[1] - t[0]);
    printf("Init Class  :\t%lf sec.\n", t[2] - t[1]);
    printf("Iteration   :\t%lf sec.\n", t[3] - t[2]);
    printf("hipMemcpy  :\t%lf sec.\n", t[4] - t[3]);
    printf("----\n");
    printf("FFT         :\t%lf sec.\n", ft);
    printf("MA          :\t%lf sec.\n", mt);
    printf("Others      :\t%lf sec.\n", t[3] - t[2] - ft - mt);
  }
#endif
} 
